#include "hip/hip_runtime.h"
#include <jansson.h>
#include <time.h>
#include "miner_cuda.h"

extern "C"{
    #include "../../include/miner.h"
    #include "../../include/json.h"
    #include "../../include/hashing.h"
}

__global__ void calculate_block_hash_cuda(long int timestamp, int id, int *previous_hash, int *transaction_hashes, int transaction_count, unsigned int nonce_low, unsigned int nonce_high, int *hash_solution, unsigned int *nonce_solution){
    int i, j;
    unsigned int nonce;
    unsigned int temp_nonce;
    unsigned int nonce_splitted[NONCE_LENGTH / GROUP_LENGTH];
    int hash[HASH_LENGTH / GROUP_LENGTH];
    int zero_count;
    
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int nonce_range = nonce_high - nonce_low;
    unsigned int low_index = nonce_low + thread_index * (nonce_range / (NUM_BLOCKS * NUM_THREADS));
    unsigned int high_index = nonce_low + (thread_index + 1) * (nonce_range / (NUM_BLOCKS * NUM_THREADS));
    
    // hashing algorithm
    for(nonce = low_index; nonce < high_index; nonce++){       
        // split nonce
        temp_nonce = nonce;
        for(i = 0; i < (NONCE_LENGTH / GROUP_LENGTH); i++){
            nonce_splitted[i] = temp_nonce % (1 << (4 * GROUP_LENGTH));
            temp_nonce /= (1 << (4 * GROUP_LENGTH));
        }
        
        // calculate block hash
        hash[0] = (long int)(timestamp + id) % (1 << (4 * GROUP_LENGTH));
        hash[0] = (hash[0] + (previous_hash[0] ^ nonce_splitted[0])) % (1 << (4 * GROUP_LENGTH));
        for(j = 0; j < transaction_count; j++){
        hash[0] = (hash[0] + (transaction_hashes[j*(HASH_LENGTH / GROUP_LENGTH)] ^ nonce_splitted[j % (NONCE_LENGTH / GROUP_LENGTH)])) % (1 << (4 * GROUP_LENGTH));
        }

        for(i = 1; i < HASH_LENGTH / GROUP_LENGTH; i++){
            hash[i] = (hash[i-1] + (previous_hash[i] ^ nonce_splitted[i % 4])) % (1 << (4 * GROUP_LENGTH));
            for(j = 0; j < transaction_count; j++){
                hash[i] = (hash[i] + (transaction_hashes[j*(HASH_LENGTH / GROUP_LENGTH) + i] ^ nonce_splitted[j % (NONCE_LENGTH / GROUP_LENGTH)])) % (1 << (4 * GROUP_LENGTH));
            }
        }

        // count zeros
        zero_count = 0;
        for(i = 0; i < HASH_LENGTH / GROUP_LENGTH; i++){
            if(hash[i] == 0){
                zero_count++;
            }
            else{
                break;
            }
        }

        // check if solution
        if(zero_count == DIFFICULTY / GROUP_LENGTH){
            for(i = 0; i < HASH_LENGTH / GROUP_LENGTH; i++){
                hash_solution[i] = hash[i];
            }
            *nonce_solution = nonce;
            break;
        }
    }
}

extern "C" int brute_force_solve_block(block_t *block){
    // cpu variables
    int *transaction_hashes;
    int *previous_hash;
    int *hash;
    unsigned int nonce;
    int i;
        
    // gpu variables
    int *transaction_hashes_gpu;
    int *previous_hash_gpu;
    int *hash_gpu;
    unsigned int *nonce_gpu;

    time_t end, start;
    double time_elapsed;
    int i_prev;

    time(&start);
    printf(BLU "\n%s" RESET, ctime(&start));
    printf(BLU "Starting Brute Force Mining of Block %d - Difficulty = %d\n" RESET, block->id, DIFFICULTY);

    // calculate transaction hashes
    transaction_hashes = calculate_transaction_hashes_1D(block->transactions, block->transaction_count);
    printf(CYN "\nCalculated Transaction Hashes\n" RESET);

    // previous hash
    previous_hash = hash_to_int_array(block->previous);

    // hash
    hash = (int*) calloc(HASH_LENGTH / GROUP_LENGTH, sizeof(int));

    // nonce
    nonce = 0;

    // allocate cuda memory
    hipMalloc(&transaction_hashes_gpu, block->transaction_count * (HASH_LENGTH / GROUP_LENGTH) * sizeof(int*));
    hipMalloc(&previous_hash_gpu, (HASH_LENGTH / GROUP_LENGTH) * sizeof(int));
    hipMalloc(&hash_gpu, (HASH_LENGTH / GROUP_LENGTH) * sizeof(int));
    hipMalloc(&nonce_gpu, sizeof(unsigned int));

    // copy from host to device
    hipMemcpy(transaction_hashes_gpu, transaction_hashes, block->transaction_count * (HASH_LENGTH / GROUP_LENGTH) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(previous_hash_gpu, previous_hash, HASH_LENGTH / GROUP_LENGTH * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(hash_gpu, hash, HASH_LENGTH / GROUP_LENGTH * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(nonce_gpu, &nonce, sizeof(unsigned int), hipMemcpyHostToDevice);

    // solve block
    printf(BLU "Starting %d mining threads\n\n" RESET, NUM_BLOCKS * NUM_THREADS);

    i_prev = -1;

    for(i = 0; i < RANGE_PARTS; i++){
        // calculate hash for nonces range
        calculate_block_hash_cuda<<< NUM_BLOCKS, NUM_THREADS >>>(block->timestamp, block->id,
            previous_hash_gpu, transaction_hashes_gpu, block->transaction_count,
            i*(UINT_MAX/RANGE_PARTS), (i + 1)*(UINT_MAX/RANGE_PARTS), hash_gpu, nonce_gpu);
        
        // wait for gpu to finish
        hipDeviceSynchronize();

        // get solution
        hipMemcpy(hash, hash_gpu, HASH_LENGTH / GROUP_LENGTH * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&nonce, nonce_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost);

        // hash string creation
        char *hash_string;
        hash_string = int_array_to_hash(hash);

        // print info
        time(&end);
        time_elapsed = difftime(end, start);

        if(time_elapsed >= MINER_CONSOLE_UPDATE_INTERVAL){
            printf(CYN "Hash Rate: %.2lf MH/s - Tried approx. %.2lf%% of all nonces\n" RESET,
                ((double)(i - i_prev) * (UINT_MAX/RANGE_PARTS)) / (1000000 * time_elapsed), (((double)(i + 1)/RANGE_PARTS) * 100));
            print_stats();

            time(&start);
        }
        
        // check if solution was found
        if(hash_ok(hash_string, DIFFICULTY)){
            printf(GRN "Found Nonce Solution: %u, Block Hash: %s!\n" RESET, nonce, hash_string);
            block->nonce = nonce;
            block->hash = hash_string;

            // cleanup
            hipFree(transaction_hashes_gpu);
            hipFree(previous_hash_gpu);
            hipFree(hash_gpu);
            hipFree(nonce_gpu);
            free(transaction_hashes);
            free(previous_hash);
            free(hash);

            return EXIT_SUCCESS;
        }        
    }          
    
    printf(RED "No Solution Found!\n" RESET);

    // cleanup
    hipFree(transaction_hashes_gpu);
    hipFree(previous_hash_gpu);
    hipFree(hash_gpu);
    hipFree(nonce_gpu);
    free(transaction_hashes);
    free(previous_hash);
    free(hash);

    return EXIT_FAILURE;
}