#include "hip/hip_runtime.h"
#include <jansson.h>
#include <time.h>
#include "miner_cuda.h"

extern "C"{
    #include "../../include/miner.h"
    #include "../../include/json.h"
    #include "../../include/hashing.h"
}

__device__ void hash_loop(unsigned int hash[], unsigned int words[]){
    int i;
    unsigned int a, b, c, d, e, f, g, h;
    unsigned int m, k, temp;

    for (i = 0; i < (HEX_LENGTH / HEX_GROUP) * 8; i++){
        a = hash[0];
        b = hash[1];
        c = hash[2];
        d = hash[3];
        e = hash[4];
        f = hash[5];
        g = hash[6];
        h = hash[7];
        
        if (i < 16){
            m = (b & c) | ((~ b) & d) | (b & (~e)) | ((~b) & (~f)) | (b & (~g)) | ((~b) & h);
            k = 1518500249; // 0x5A827999
        }
        else if (i < 32){
            m = b ^ c ^ d ^ e ^ f ^ g ^ h;
            k = 1859775393; // 0x6ED9EBA1
        }
        else if (i < 48){
            m = (b & c) | (b & d) | (b & e) | (b & f) | (b & g) | (b & h);
            k = 2400959708; // 0x8F1BBCDC
        }
        else{
            m = b ^ c ^ d ^ e ^ f ^ g ^ h;
            k = 3395469782; // 0xCA62C1D6
        }

        temp = ((a << 5) | (a >> 27)) ^ (m & k) ^ words[i];

        h = g;
        g = (f << 29) | (f >> 3);
        f = e;
        e = d;
        d = c;
        c = (b << 13) | (b >> 19);
        b = a;
        a = temp;

        hash[0] ^= a;
        hash[1] ^= b;
        hash[2] ^= c;
        hash[3] ^= d;
        hash[4] ^= e;
        hash[5] ^= f;
        hash[6] ^= g;
        hash[7] ^= h;
    }
}

__global__ void calculate_block_hash_cuda(unsigned long int timestamp, unsigned int id, unsigned int *previous_hash, unsigned int *transaction_hashes, int transaction_count, unsigned int nonce_low, unsigned int nonce_high, unsigned int *hash_solution, unsigned int *nonce_solution){
    int i, j;
    unsigned int nonce;
    unsigned int hash[HEX_LENGTH / HEX_GROUP];
    unsigned int words[(HEX_LENGTH / HEX_GROUP) * 8];
    
    unsigned int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int nonce_range = nonce_high - nonce_low;
    unsigned int low_index = nonce_low + thread_index * (nonce_range / (NUM_BLOCKS * NUM_THREADS));
    unsigned int high_index = nonce_low + (thread_index + 1) * (nonce_range / (NUM_BLOCKS * NUM_THREADS));
    
    // hashing algorithm
    for(nonce = low_index; nonce < high_index; nonce++){  

        /* Step 1 - Init Hash with Previous Block's Hash */
        for (i = 0; i < (int)(HEX_LENGTH / HEX_GROUP); i++){
            hash[i] = previous_hash[i];
        }

        /* Step 2 - Add Block Info */
        words[0] = nonce;
        words[1] = (timestamp >> 32) % UINT_MAX;
        words[2] = timestamp % UINT_MAX;
        words[3] = id;
        
        for (i = 4; i < (HEX_LENGTH / HEX_GROUP) * 8; i++){
            words[i] = ((words[i - 2] ^ words[i - 4]) << 3) | ((words[i - 2] ^ words[i - 4]) >> 29);
        }

        hash_loop(hash, words);

        /* Step 3 - Add Transaction Hashes */
        for (i = 0; i < transaction_count; i++){
            for (j = 0; j < (HEX_LENGTH / HEX_GROUP); j++){
                words[j] = transaction_hashes[i * (HEX_LENGTH / HEX_GROUP) + j];
            }
            
            for (j = (HEX_LENGTH / HEX_GROUP); j < (HEX_LENGTH / HEX_GROUP) * 8; j++){
                words[j] = ((words[j - 2] ^ words[j - 4]) << 3) | ((words[j - 2] ^ words[j - 4]) >> 29);
            }

            hash_loop(hash, words);
        }
        
        // check if solution
        if(DIFFICULTY == 8){
            if(hash[0] == 0){
                for(i = 0; i < HEX_LENGTH / HEX_GROUP; i++){
                    hash_solution[i] = hash[i];
                }
                *nonce_solution = nonce;
                break;
            }
        }
        else if(hash[0] <= (1 << (32 - DIFFICULTY*4) - 1)){
            for(i = 0; i < HEX_LENGTH / HEX_GROUP; i++){
                hash_solution[i] = hash[i];
            }
            *nonce_solution = nonce;
            break;
        }
    }
}

extern "C" int brute_force_solve_block(block_t *block){
    // cpu variables
    unsigned int *transaction_hashes;
    unsigned int *previous_hash;
    unsigned int *hash;
    unsigned int nonce;
    long int i;
        
    // gpu variables
    unsigned int *transaction_hashes_gpu;
    unsigned int *previous_hash_gpu;
    unsigned int *hash_gpu;
    unsigned int *nonce_gpu;

    time_t end, start;
    double time_elapsed;
    long int i_prev;

    time(&start);
    printf(BLU "\n%s" RESET, ctime(&start));
    printf(BLU "Starting Brute Force Mining of Block %d - Difficulty = %d\n" RESET, block->id, DIFFICULTY);

    // calculate transaction hashes
    transaction_hashes = calculate_transaction_hashes_1D(block->transactions, block->transaction_count);
    printf(CYN "\nCalculated Transaction Hashes\n" RESET);

    // previous hash
    previous_hash = hash_to_int_array(block->previous);

    // hash
    hash = (unsigned int*) calloc(HEX_LENGTH / HEX_GROUP, sizeof(unsigned int));

    // nonce
    nonce = 0;

    // allocate cuda memory
    hipMalloc(&transaction_hashes_gpu, block->transaction_count * (HEX_LENGTH / HEX_GROUP) * sizeof(unsigned int));
    hipMalloc(&previous_hash_gpu, (HEX_LENGTH / HEX_GROUP) * sizeof(unsigned int));
    hipMalloc(&hash_gpu, (HEX_LENGTH / HEX_GROUP) * sizeof(unsigned int));
    hipMalloc(&nonce_gpu, sizeof(unsigned int));

    // copy from host to device
    hipMemcpy(transaction_hashes_gpu, transaction_hashes, block->transaction_count * (HEX_LENGTH / HEX_GROUP) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(previous_hash_gpu, previous_hash, HEX_LENGTH / HEX_GROUP * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(hash_gpu, hash, HEX_LENGTH / HEX_GROUP * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(nonce_gpu, &nonce, sizeof(unsigned int), hipMemcpyHostToDevice);

    // solve block
    printf(BLU "Starting %d mining threads\n\n" RESET, NUM_BLOCKS * NUM_THREADS);

    i_prev = -1;

    for(i = 0; i < RANGE_PARTS; i++){
        // calculate hash for nonces range
        calculate_block_hash_cuda<<< NUM_BLOCKS, NUM_THREADS >>>(block->timestamp, block->id,
            previous_hash_gpu, transaction_hashes_gpu, block->transaction_count,
            i*(UINT_MAX/RANGE_PARTS), (i + 1)*(UINT_MAX/RANGE_PARTS), hash_gpu, nonce_gpu);
        
        // wait for gpu to finish
        hipDeviceSynchronize();

        // get solution
        hipMemcpy(hash, hash_gpu, HEX_LENGTH / HEX_GROUP * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(&nonce, nonce_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost);

        // hash string creation
        char *hash_string;
        hash_string = int_array_to_hash(hash);

        // print info
        time(&end);
        time_elapsed = difftime(end, start);
        
        if(time_elapsed >= MINER_CONSOLE_UPDATE_INTERVAL){
            printf(CYN "Hash Rate: %.2lf MH/s - Tried approx. %.2lf%% of all nonces\n" RESET,
                ((double)(i - i_prev) * (UINT_MAX/RANGE_PARTS)) / (1000000 * time_elapsed), (((double)(i + 1)/RANGE_PARTS) * 100));
            print_stats();
            time(&start);
            i_prev = i;

            time(&start);
        }
        
        // check if solution was found
        if(hash_ok(hash_string, DIFFICULTY)){
            printf(CYN "Hash Rate: %.2lf MH/s - Tried approx. %.2lf%% of all nonces\n" RESET,
                ((double)(i - i_prev) * (UINT_MAX/RANGE_PARTS)) / (1000000 * time_elapsed), (((double)(i + 1)/RANGE_PARTS) * 100));
            print_stats();
            time(&start);
            i_prev = i;

            printf(GRN "Found Nonce Solution: %u, Block Hash: %s!\n" RESET, nonce, hash_string);
            block->nonce = nonce;
            block->hash = hash_string;

            // cleanup
            hipFree(transaction_hashes_gpu);
            hipFree(previous_hash_gpu);
            hipFree(hash_gpu);
            hipFree(nonce_gpu);
            free(transaction_hashes);
            free(previous_hash);
            free(hash);

            return EXIT_SUCCESS;
        }
    }          
    
    printf(RED "No Solution Found!\n" RESET);

    // cleanup
    hipFree(transaction_hashes_gpu);
    hipFree(previous_hash_gpu);
    hipFree(hash_gpu);
    hipFree(nonce_gpu);
    free(transaction_hashes);
    free(previous_hash);
    free(hash);

    return EXIT_FAILURE;
}
